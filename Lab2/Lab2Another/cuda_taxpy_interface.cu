#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "omp.h"
#include "cuda_taxpy_template.h"

//Cuda version

template <typename T>
__global__ void taxpy_kernel(int n, T* X, int Xinc, T* Y, int Yinc, T alpfa){
    int op_nom = std::ceil(((double)(n) / (double)(max(Xinc,Yinc))));
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < op_nom)
        Y[i*Yinc]+=alpfa*X[i*Xinc];
}

template <typename T>
double cuda_taxpy(int n, T* X, int Xinc, T* Y, int Yinc, T alpfa, int blocksPerGrid, int threadsPerBlock){
    hipError_t err = hipSuccess;

    //memory allocation
    T *gpuX, *gpuY;
    err = hipMalloc((void**)&gpuX, n*sizeof(T));
    if (err != hipSuccess){
        printf("gpuX memory allocation error. ");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&gpuY, n*sizeof(T));
    if (err != hipSuccess){
        printf("gpuY memory allocation error. ");
        exit(EXIT_FAILURE);
    }

    //memory relocation Host to device
    err = hipMemcpy(gpuX, X, n*sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("gpuX memory relocation error. Host to device.");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(gpuY, X, n*sizeof(T), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        printf("gpuY memory relocation error. Host to device.");
        exit(EXIT_FAILURE);
    }

    //Launch kernel and mark the time
    double start = omp_get_wtime();
    taxpy_kernel<T><<<blocksPerGrid, threadsPerBlock>>>(n, gpuX, Xinc, gpuY, Yinc, alpfa);
    hipDeviceSynchronize();
    double end = omp_get_wtime();
    
    //memory relocation Device to host
    err = hipMemcpy(Y, gpuY, n*sizeof(T), hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        printf("gpuY memory relocation error. Device to host.");
        exit(EXIT_FAILURE);
    }
    //freeing memory 
    err = hipFree(gpuX);
    if (err != hipSuccess){
        printf("gpuX destruction error. ");
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(gpuY);
    if (err != hipSuccess){
        printf("gpuY destruction error. ");
        exit(EXIT_FAILURE);
    }

    //time return
    return end - start;
}


/* ????? To correct ????? */
double cuda_daxpy(int n, double* X, int Xinc, double* Y, int Yinc, double alpfa, int blocksPerGrid, int threadsPerBlock){
    return cuda_taxpy<double> (n, X, Xinc,Y, Yinc, alpfa, blocksPerGrid, threadsPerBlock);
}

double cuda_faxpy(int n, float* X, int Xinc, float* Y, int Yinc, float alpfa, int blocksPerGrid, int threadsPerBlock){
    return cuda_taxpy<float> (n, X, Xinc, Y, Yinc, alpfa, blocksPerGrid, threadsPerBlock);
}
/*------------------------*/
